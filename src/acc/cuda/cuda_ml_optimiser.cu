#include "hip/hip_runtime.h"
#undef ALTCPU
#include <sys/time.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <ctime>
#include <vector>
#include <iostream>
#include "src/ml_optimiser.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "src/acc/acc_ptr.h"
#include "src/acc/acc_projector.h"
#include "src/acc/acc_backprojector.h"
#include "src/acc/acc_backprojector_impl.h"
#include "src/acc/acc_projector_plan.h"
#include "src/acc/cuda/cuda_benchmark_utils.h"
#include "src/acc/cuda/cuda_kernels/helper.cuh"
#include "src/acc/cuda/cuda_kernels/diff2.cuh"
#include "src/acc/cuda/cuda_kernels/wavg.cuh"
#include "src/acc/cuda/cuda_mem_utils.h"
#include "src/acc/cuda/cuda_fft.h"
#include "src/acc/data_types.h"
#include "src/complex.h"
#include "src/helix.h"
#include "src/error.h"
#include <fstream>
#include "src/parallel.h"
#include <signal.h>
#include <map>

#ifdef CUDA_FORCESTL
#include "src/acc/cuda/cuda_utils_stl.cuh"
#else
#include "src/acc/cuda/cuda_utils_cub.cuh"
#endif

#include "src/acc/utilities.h"
#include "src/acc/utilities_impl.h"

#include "src/acc/acc_ml_optimiser.h"
#include "src/acc/cuda/cuda_ml_optimiser.h"
#include "src/acc/acc_helper_functions.h"
#include "src/acc/acc_ml_optimiser_impl.h"

// -------------------------------  Some explicit template instantiations
template __global__ void CudaKernels::cuda_kernel_translate2D<XFLOAT>(
    XFLOAT const*, XFLOAT*, int, int, int, int, int
);

template __global__ void CudaKernels::cuda_kernel_translate3D<XFLOAT>(
    XFLOAT const*, XFLOAT*, int, int, int, int, int, int, int
);

template __global__ void cuda_kernel_multi<XFLOAT>(
    XFLOAT const*, XFLOAT*, XFLOAT, int
);

template __global__ void CudaKernels::cuda_kernel_multi<XFLOAT>(
    XFLOAT*, XFLOAT, int
);

template __global__ void cuda_kernel_multi<XFLOAT>(
    XFLOAT const*, XFLOAT const*, XFLOAT*, XFLOAT, int
);

// ----------------------------------------------------------------------

// High-level CUDA objects

size_t MlDeviceBundle::checkFixedSizedObjects(int shares) {
    int devCount;
    HANDLE_ERROR(hipGetDeviceCount(&devCount));
    if (device_id >= devCount) {
        CRITICAL(ERR_GPUID);
    }

    HANDLE_ERROR(hipSetDevice(device_id));

    size_t free(0), total(0);
    DEBUG_HANDLE_ERROR(hipMemGetInfo(&free, &total));
    float margin(1.05);
    const size_t BoxLimit = pow(free / (margin * 2.5 * sizeof(XFLOAT) * ((float) shares)),(1 / 3.0)) / ((float) baseMLO->mymodel.padding_factor);
    // size_t BytesNeeded = ((float) shares) * margin * 2.5 * sizeof(XFLOAT) * pow((baseMLO->mymodel.ori_size * baseMLO->mymodel.padding_factor),3);
    return BoxLimit;
}

void MlDeviceBundle::setupFixedSizedObjects() {
    int devCount;
    HANDLE_ERROR(hipGetDeviceCount(&devCount));
    if (device_id >= devCount) {
        //std::cerr << " using device_id=" << device_id << " (device no. " << device_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
        CRITICAL(ERR_GPUID);
    } else {
        HANDLE_ERROR(hipSetDevice(device_id));
    }

    // Can we pre-generate projector plan and corresponding euler matrices for all particles
    generateProjectionPlanOnTheFly = baseMLO->do_skip_align || baseMLO->do_skip_rotate || baseMLO->do_auto_refine || baseMLO->mymodel.orientational_prior_mode != NOPRIOR;

    const unsigned nr_proj  = baseMLO->mymodel.PPref.size();
    const unsigned nr_bproj = baseMLO->wsum_model.BPref.size();

    projectors.resize(nr_proj);
    backprojectors.resize(nr_bproj);

    /*======================================================
                  PROJECTOR AND BACKPROJECTOR
    ======================================================*/

    for (int i = 0; i < nr_proj; i++) {
        const auto& pp = baseMLO->mymodel.PPref[i];
        projectors[i].setMdlDim(
            pp.data.xdim, pp.data.ydim, pp.data.zdim,
            pp.data.yinit, pp.data.zinit,
            pp.r_max, pp.padding_factor
        );
        projectors[i].initMdl(pp.data.data);
    }

    for (int i = 0; i < nr_bproj; i++) {
        const auto& bp = baseMLO->wsum_model.BPref[i];
        backprojectors[i].setMdlDim(
            bp.data.xdim, bp.data.ydim, bp.data.zdim,
            bp.data.yinit, bp.data.zinit,
            bp.r_max, bp.padding_factor
        );
        backprojectors[i].initMdl();
    }

    /*======================================================
                        CUSTOM ALLOCATOR
    ======================================================*/

    int memAlignmentSize;
    hipDeviceGetAttribute(&memAlignmentSize, hipDeviceAttributeTextureAlignment, device_id);
    allocator = new CudaCustomAllocator(0, memAlignmentSize);
}

void MlDeviceBundle::setupTunableSizedObjects(size_t allocationSize) {
    int devCount;
    HANDLE_ERROR(hipGetDeviceCount(&devCount));
    if (device_id >= devCount) {
        //std::cerr << " using device_id=" << device_id << " (device no. " << device_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
        CRITICAL(ERR_GPUID);
    } else {
        HANDLE_ERROR(hipSetDevice(device_id));
    }

    /*======================================================
                        CUSTOM ALLOCATOR
    ======================================================*/
    #ifdef DEBUG_CUDA
    printf("DEBUG: Total GPU allocation size set to %zu MB on device id %d.\n", allocationSize / (1000*1000), device_id);
    #endif
    #ifndef CUDA_NO_CUSTOM_ALLOCATION
    allocator->resize(allocationSize);
    #endif

    /*======================================================
                        PROJECTION PLAN
    ======================================================*/

    const unsigned nr_models = baseMLO->mymodel.nr_classes;
    coarseProjectionPlans.resize(nr_models, allocator);

    if (generateProjectionPlanOnTheFly) return;
    // Otherwise, if doing predefined projector plan:
    for (int iclass = 0; iclass < nr_models; iclass++) {
        // Skip insignificant classes
        if (baseMLO->mymodel.pdf_class[iclass] <= 0.0) continue;
        std::vector<int> exp_pointer_dir_nonzeroprior;
        std::vector<int> exp_pointer_psi_nonzeroprior;
        std::vector<RFLOAT> exp_directions_prior;
        std::vector<RFLOAT> exp_psi_prior;

        const long unsigned itrans_max = baseMLO->sampling.NrTranslationalSamplings() - 1;
        const long unsigned nr_idir    = baseMLO->sampling.NrDirections();
        const long unsigned nr_ipsi    = baseMLO->sampling.NrPsiSamplings();

        coarseProjectionPlans[iclass].setup(
            baseMLO->sampling,
            exp_directions_prior,
            exp_psi_prior,
            exp_pointer_dir_nonzeroprior,
            exp_pointer_psi_nonzeroprior,
            nullptr,  // Mcoarse_significant
            baseMLO->mymodel.pdf_class,
            baseMLO->mymodel.pdf_direction,
            nr_idir, nr_ipsi,
            0 /*idir_min*/, nr_idir - 1 /*idir_max*/,
            0 /*ipsi_min*/, nr_ipsi - 1 /*ipsi_max*/,
            0 /*itrans_min*/, itrans_max,
            0 /*current_oversampling*/, 1 /*nr_oversampled_rot*/,
            iclass,
            true,  // coarse
            !IS_NOT_INV,
            baseMLO->do_skip_align,
            baseMLO->do_skip_rotate,
            baseMLO->mymodel.orientational_prior_mode
        );
    }
};

void MlOptimiserCuda::resetData() {
    int devCount;
    HANDLE_ERROR(hipGetDeviceCount(&devCount));
    if (device_id >= devCount) {
        //std::cerr << " using device_id=" << device_id << " (device no. " << device_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
        CRITICAL(ERR_GPUID);
    } else {
        HANDLE_ERROR(hipSetDevice(device_id));
    }

    classStreams.resize(baseMLO->mymodel.nr_classes, nullptr);
    for (hipStream_t& stream: classStreams) {
        HANDLE_ERROR(hipStreamCreate(&stream));
        // HANDLE_ERROR(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    }

    transformer1.clear();
    transformer2.clear();
};

void MlOptimiserCuda::doThreadExpectationSomeParticles(int thread_id) {
    #ifdef TIMING
    // Only time one thread
    if (thread_id == 0)
    baseMLO->timer.tic(baseMLO->TIMING_ESP_THR);
    #endif
	// CTOC(cudaMLO->timer, "interParticle");

    int devCount;
    HANDLE_ERROR(hipGetDeviceCount(&devCount));
    if (device_id >= devCount) {
        // std::cerr << " using device_id=" << device_id << " (device no. " << device_id+1 << ") which is higher than the available number of devices=" << devCount << std::endl;
        CRITICAL(ERR_GPUID);
    } else {
        DEBUG_HANDLE_ERROR(hipSetDevice(device_id));
    }
    // std::cerr << " calling on device " << device_id << std::endl;
    // put mweight allocation here
    size_t first_ipart = 0, last_ipart = 0;

    while (baseMLO->exp_ipart_ThreadTaskDistributor->getTasks(first_ipart, last_ipart)) {
        CTICTOC(timer, "oneTask", ({
        for (long unsigned ipart = first_ipart; ipart <= last_ipart; ipart++) {
            #ifdef TIMING
            // Only time one thread
            if (thread_id == 0)
            baseMLO->timer.tic(baseMLO->TIMING_ESP_DIFF2_A);
            #endif

            AccPtrFactory<> ptrFactory (allocator, hipStreamPerThread);
            accDoExpectationOneParticle<MlOptimiserCuda>(this, baseMLO->exp_my_first_part_id + ipart, thread_id, ptrFactory);

        }
        }))
    }

    // CTIC(cudaMLO->timer, "interParticle");
    // exit(0);

    #ifdef TIMING
    // Only time one thread
    if (thread_id == 0)
    baseMLO->timer.toc(baseMLO->TIMING_ESP_THR);
    #endif
}

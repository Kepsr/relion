#include "hip/hip_runtime.h"
/**
 * WARNING:
 * This file (src/gpu_utils/cuda_autopicker.cu) is not referenced by any other,
 * and is a doublet of src/acc/cuda/cuda_autopicker.cu
 *
 * ('Doublet' in the sense that it implements functions
 * of the same name as those implemented in src/acc/cuda/cuda_autopicker.cu,
 * but those functions may not behave exactly the same.)
*/
#include <sys/time.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <ctime>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <signal.h>
#include "src/gpu_utils/cuda_autopicker.h"

#include "src/gpu_utils/cuda_mem_utils.h"
#include "src/gpu_utils/cuda_projector.h"
#include "src/gpu_utils/cuda_settings.h"
#include "src/gpu_utils/cuda_benchmark_utils.h"
#include "src/gpu_utils/cuda_helper_functions.cuh"
#include "src/gpu_utils/cuda_fft.h"

#include "src/macros.h"
#include "src/error.h"

#ifdef CUDA_FORCESTL
#include "src/gpu_utils/cuda_utils_stl.cuh"
#else
#include "src/gpu_utils/cuda_utils_cub.cuh"
#endif

// Z-score of x given mean mu and standard deviation sigma
inline RFLOAT Z(RFLOAT x, RFLOAT mu, RFLOAT sigma) {
    return (x - mu) / sigma;
}

AutoPickerCuda::AutoPickerCuda(AutoPicker *basePicker, int dev_id, const char * timing_fnm):
    node(NULL), basePckr(basePicker),
    allocator(new CudaCustomAllocator(0, 1)),
    micTransformer(0, allocator),
    cudaTransformer1(0, allocator),
    #ifdef TIMING_FILES
    timer(timing_fnm),
    #endif
    cudaTransformer2(0, allocator) {

    cudaProjectors.resize(basePckr->Mrefs.size());
    have_warned_batching = false;
    /*======================================================
                        DEVICE SETTINGS
    ======================================================*/
    device_id = dev_id;
    int devCount;
    HANDLE_ERROR(hipGetDeviceCount(&devCount));

    if (dev_id >= devCount) {
        // std::cerr << " using device_id=" << dev_id << " (device no. " << dev_id + 1 << ") which is higher than the available number of devices=" << devCount << std::endl;
        CRITICAL(ERR_GPUID);
    } else {
        HANDLE_ERROR(hipSetDevice(dev_id));
    }
};

AutoPickerCuda::AutoPickerCuda(AutoPickerMpi *basePicker, int dev_id, const char *timing_fnm):
    basePckr(basePicker),
    allocator(new CudaCustomAllocator(0, 1)),
    micTransformer(0, allocator),
    cudaTransformer1(0, allocator),
    #ifdef TIMING_FILES
    timer(timing_fnm),
    #endif
    cudaTransformer2(0, allocator) {

    node = basePicker->getNode();
    /// BUG: class "MpiNode" has no member "isMaster"
    basePicker->verb = node->isMaster() ? 1 : 0;

    cudaProjectors.resize(basePckr->Mrefs.size());
    have_warned_batching = false;
    /*======================================================
                        DEVICE SETTINGS
    ======================================================*/
    device_id = dev_id;
    int devCount;
    HANDLE_ERROR(hipGetDeviceCount(&devCount));

    if (dev_id >= devCount) {
        // std::cerr << " using device_id=" << dev_id << " (device no. " << dev_id + 1 << ") which is higher than the available number of devices=" << devCount << std::endl;
        CRITICAL(ERR_GPUID);
    } else {
        HANDLE_ERROR(hipSetDevice(dev_id));
    }
};

void AutoPickerCuda::run() {
    long int my_first_micrograph, my_last_micrograph, my_nr_micrographs;
    if (node != NULL) {
        // Each node does part of the work
        divide_equally(basePckr->fn_micrographs.size(), node->size, node->rank, my_first_micrograph, my_last_micrograph);
    } else {
        my_first_micrograph = 0;
        my_last_micrograph = basePckr->fn_micrographs.size() - 1;
    }
    my_nr_micrographs = my_last_micrograph - my_first_micrograph + 1;

    int barstep;
    if (basePckr->verb > 0) {
        std::cout << " Autopicking ..." << std::endl;
        init_progress_bar(my_nr_micrographs);
        barstep = std::max(1, (int) my_nr_micrographs / 60);
    }

    if (!basePckr->do_read_fom_maps) {
        CTICTOC(timer, "setupProjectors", ({
        for (int iref = 0; iref < basePckr->Mrefs.size(); iref++) {
            cudaProjectors[iref].setMdlDim(
                basePckr->PPref[iref].data.xdim,
                basePckr->PPref[iref].data.ydim,
                basePckr->PPref[iref].data.zdim,
                basePckr->PPref[iref].data.yinit,
                basePckr->PPref[iref].data.zinit,
                basePckr->PPref[iref].r_max,
                basePckr->PPref[iref].padding_factor
            );
            cudaProjectors[iref].initMdl(&basePckr->PPref[iref].data.data[0]);
        }
        }))
    }

    FileName fn_olddir = "";

    for (long int imic = my_first_micrograph; imic <= my_last_micrograph; imic++) {
        if (basePckr->verb > 0 && imic % barstep == 0)
            progress_bar(imic);


        // Check new-style outputdirectory exists and make it if not!
        FileName fn_dir = basePckr->getOutputRootName(basePckr->fn_micrographs[imic]);
        fn_dir = fn_dir.beforeLastOf("/");
        if (fn_dir != fn_olddir) {
            // Make a Particles directory
            system(("mkdir -p " + fn_dir).c_str());
            fn_olddir = fn_dir;
        }
        #ifdef TIMING
        basePckr->timer.tic(basePckr->TIMING_A5);
        #endif
        autoPickOneMicrograph(basePckr->fn_micrographs[imic], imic);
    }
    /// BUG: Should this toc be inside the above loop?
        #ifdef TIMING
        basePckr->timer.toc(basePckr->TIMING_A5);
        #endif
    if (basePckr->verb > 0)
        progress_bar(my_nr_micrographs);

    hipDeviceReset();

}

void AutoPickerCuda::calculateStddevAndMeanUnderMask(
    CudaGlobalPtr<CUDACOMPLEX> &d_Fmic, CudaGlobalPtr<CUDACOMPLEX> &d_Fmic2, CudaGlobalPtr<CUDACOMPLEX> &d_Fmsk,
    int nr_nonzero_pixels_mask,
    CudaGlobalPtr<XFLOAT> &d_Mstddev, CudaGlobalPtr<XFLOAT> &d_Mmean,
    size_t x, size_t y, size_t mic_size, size_t workSize
) {
    cudaTransformer2.setSize(workSize, workSize, 1);

    deviceInitValue(d_Mstddev, (XFLOAT) 0.0);

    RFLOAT normfft = (RFLOAT) (mic_size * mic_size) / (RFLOAT) nr_nonzero_pixels_mask;

    CudaGlobalPtr<CUDACOMPLEX> d_Fcov(d_Fmic.getAllocator());
    d_Fcov.device_alloc(d_Fmic.getSize());

    CTICTOC(timer, "PRE-multi_0", ({
    int Bsize = ceilf((float) d_Fmic.size / (float) BLOCK_SIZE);
    cuda_kernel_convol_B<<<Bsize, BLOCK_SIZE>>>(
        ~d_Fmic, ~d_Fmsk, ~d_Fcov, d_Fmic.getSize()
    );
    LAUNCH_HANDLE_ERROR(hipGetLastError());
    }))

    CTICTOC(timer, "PRE-window_0", ({
    windowFourierTransform2(
        d_Fcov,
        cudaTransformer2.fouriers,
        x, y, 1,
        workSize / 2 + 1, workSize, 1
    );
    }))

    CTICTOC(timer, "PRE-Transform_0", ({ cudaTransformer2.backward(); }))

    Bsize = ceilf((float) cudaTransformer2.reals.size / (float) BLOCK_SIZE);
    cuda_kernel_multi<<<Bsize, BLOCK_SIZE>>>(
        cudaTransformer2.reals.d_ptr,
        cudaTransformer2.reals.d_ptr,
        (XFLOAT) normfft,
        cudaTransformer2.reals.size
    );
    LAUNCH_HANDLE_ERROR(hipGetLastError());

    CTICTOC(timer, "PRE-multi_1", ({
    cuda_kernel_multi<<<Bsize, BLOCK_SIZE>>>(
        cudaTransformer2.reals.d_ptr,
        cudaTransformer2.reals.d_ptr,
        d_Mstddev.d_ptr,
        (XFLOAT) -1,
        cudaTransformer2.reals.size
    );
    LAUNCH_HANDLE_ERROR(hipGetLastError());
    }))

    CTICTOC(timer, "PRE-CenterFFT_0", ({
    runCenterFFT(
        cudaTransformer2.reals,
        (int) cudaTransformer2.xSize,
        (int) cudaTransformer2.ySize,
        false,
        1
    );
    }))

    cudaTransformer2.reals.cp_on_device(d_Mmean); //TODO remove the need for this

    CTICTOC(timer, "PRE-multi_2", ({
    Bsize = ((int) ceilf((float) d_Fmsk.size / (float) BLOCK_SIZE));
    cuda_kernel_convol_A<<<Bsize, BLOCK_SIZE>>>(
        ~d_Fmsk, ~d_Fmic2, ~d_Fcov, d_Fmsk.size
    );
    LAUNCH_HANDLE_ERROR(hipGetLastError());
    }))

    CTICTOC(timer, "PRE-window_1", ({
    windowFourierTransform2(
        d_Fcov,
        cudaTransformer2.fouriers,
        x, y, 1,
        workSize / 2 + 1, workSize, 1
    );
    }))

    CTICTOC(timer, "PRE-Transform_1", ({ cudaTransformer2.backward(); }))

    CTICTOC(timer, "PRE-multi_3", ({
    Bsize = ceilf((float) d_Mstddev.size / (float) BLOCK_SIZE);
    cuda_kernel_finalizeMstddev<<<Bsize, BLOCK_SIZE>>>(
        d_Mstddev.d_ptr,
        cudaTransformer2.reals.d_ptr,
        normfft,
        d_Mstddev.size
    );
    LAUNCH_HANDLE_ERROR(hipGetLastError());
    }))

    CTICTOC(timer, "PRE-CenterFFT_1", ({
    runCenterFFT(d_Mstddev, (int) workSize, (int) workSize, false, 1);
    }))

}

void AutoPickerCuda::autoPickOneMicrograph(FileName &fn_mic, long int imic) {
    Image<RFLOAT> Imic;
    MultidimArray<Complex> Faux, Faux2, Fmic;
    MultidimArray<RFLOAT> Maux, Mstddev, Mccf_best, Mpsi_best, Fctf, Mccf_best_combined;
    MultidimArray<int> Mclass_best_combined;

    CudaGlobalPtr<XFLOAT> d_Mccf_best(basePckr->workSize * basePckr->workSize, allocator);
    CudaGlobalPtr<XFLOAT> d_Mpsi_best(basePckr->workSize * basePckr->workSize, allocator);
    d_Mccf_best.device_alloc();
    d_Mpsi_best.device_alloc();

    // Always use the same random seed
    init_random_generator(basePckr->random_seed + imic);

    RFLOAT sum_ref_under_circ_mask, sum_ref2_under_circ_mask;
    int my_skip_side = basePckr->autopick_skip_side + basePckr->particle_size / 2;

    int Npsi = 360 / basePckr->psi_sampling;

    int min_distance_pix = round(basePckr->min_particle_distance / basePckr->angpix);
    XFLOAT scale = (XFLOAT) basePckr->workSize / (XFLOAT) basePckr->micrograph_size;

    // Read in the micrograph
    {
    ifdefTIMING(TicToc tt (basePckr->timer.tic, basePckr->TIMING_A6);)
    CTICTOC(timer, "readMicrograph", ({ Imic.read(fn_mic); }))
    CTICTOC(timer, "setXmippOrigin_0", ({ Imic().setXmippOrigin(); }))
    }

    // Let's just check the square size again....
    RFLOAT my_xsize = XSIZE(Imic());
    RFLOAT my_ysize = YSIZE(Imic());
    RFLOAT my_size = std::max(my_xsize, my_ysize);

    if (
        my_xsize != basePckr->micrograph_xsize ||
        my_ysize != basePckr->micrograph_ysize ||
        my_size  != basePckr->micrograph_size
    ) {
        Imic().printShape();
        std::cerr << " micrograph_size= " << basePckr->micrograph_size << " micrograph_xsize= " << basePckr->micrograph_xsize << " micrograph_ysize= " << basePckr->micrograph_ysize << std::endl;
        REPORT_ERROR("AutoPicker::autoPickOneMicrograph ERROR: No differently sized micrographs are allowed in one run, sorry you will have to run separately for each size...");
    }

    if (!basePckr->do_read_fom_maps) {
        CTICTOC(timer, "setSize_micTr", ({
        micTransformer.setSize(basePckr->micrograph_size, basePckr->micrograph_size, 1, 1);
        }))

        CTICTOC(timer, "setSize_cudaTr", ({
        cudaTransformer1.setSize(basePckr->workSize,basePckr->workSize, 1, Npsi, FFTW_BACKWARD);
        }))
    }
    HANDLE_ERROR(hipDeviceSynchronize());

    if (cudaTransformer1.batchSize.size() > 1 && !have_warned_batching) {
        have_warned_batching = true;
        std::cerr << std::endl << "*-----------------------------WARNING------------------------------------------------*" << std::endl;
        std::cerr              << "With the current settings the GPU memory is imposing a soft limit on your performace," << std::endl;
        std::cerr              << "since one or more micrographs has to use (at least " << cudaTransformer1.batchSize.size() << ") batches of orientations to " << std::endl;
        std::cerr              << "achieve the total requested " << Npsi << " orientations. Consider using" << std::endl;
        std::cerr              << "\t higher --ang" << std::endl;
        std::cerr              << "\t harder --shrink" << std::endl;
        std::cerr              << "\t higher --lowpass with --shrink 0" << std::endl;
        std::cerr              << "*------------------------------------------------------------------------------------*" << std::endl;
    }

    {
    ifdefTIMING(TicToc tt (basePckr->timer, basePckr->TIMING_A7);)
    const auto stats = [&] () -> Stats<RFLOAT> {
        CTICTOC(timer, "computeStats", ({
        // Set mean to zero and stddev to 1 to prevent numerical problems with one-sweep stddev calculations....
        return computeStats(Imic());
        }))}();
        avg0 = stats.avg;
        stddev0 = stats.stddev;
    }

    CTICTOC(timer, "middlePassFilter", ({
    for (long int n = 0; n < Imic().size(); n++) {
        // Remove pixel values that are too far away from the mean
        if (abs(Z(Imic()[n], avg0, stddev0)) > basePckr->outlier_removal_zscore)
            Imic()[n] = avg0;

        Imic()[n] = Z(Imic()[n], avg0, stddev0);
    }
    }))

    if (basePckr->micrograph_xsize != basePckr->micrograph_ysize) {
        // Window non-square micrographs to be a square with the largest side
        CTICTOC(timer, "rewindow", ({ rewindow(Imic, basePckr->micrograph_size); }))

        // Fill region outside the original window with white Gaussian noise to prevent all-zeros in Mstddev
        CTICTOC(timer, "gaussNoiseOutside", ({
        FOR_ALL_ELEMENTS_IN_ARRAY2D(Imic(), i, j) {
            if (
                j < Xmipp::init(basePckr->micrograph_ysize) ||
                j > Xmipp::last(basePckr->micrograph_ysize) ||
                i < Xmipp::init(basePckr->micrograph_xsize) ||
                i > Xmipp::last(basePckr->micrograph_xsize)
            ) {
                Imic().elem(i, j) = rnd_gaus(0.0, 1.0);
            }
        }
        }))
    }

    #ifdef TIMING
    basePckr->timer.tic(basePckr->TIMING_A8);
    #endif
    CTICTOC(timer, "CTFread", ({
    // Read in the CTF information if needed
    if (basePckr->do_ctf) {
        // Search for this micrograph in the metadata table
        for (long int index : basePckr->MDmic) {
            FileName fn_tmp = basePckr->MDmic.getValue(EMDL::MICROGRAPH_NAME);
            if (fn_tmp == fn_mic) {
                CTF ctf = CTF(basePckr->MDmic);
                Fctf = CtfHelper::getFftwImage(
                    ctf,
                    basePckr->workSize / 2 + 1, basePckr->workSize, 
                    basePckr->micrograph_size, basePckr->micrograph_size,
                    basePckr->angpix,
                    NULL,  // No ObservationModel
                    false, false, basePckr->intact_ctf_first_peak, true
                );
                break;
            }
        }
    }
    }))
    #ifdef TIMING
    basePckr->timer.toc(basePckr->TIMING_A8);
    #endif

    #ifdef TIMING
    basePckr->timer.tic(basePckr->TIMING_A9);
    #endif
    CTICTOC(timer, "mccfResize", ({
    Mccf_best.resize(basePckr->workSize,basePckr->workSize);
    }))

    CTICTOC(timer, "mpsiResize", ({
    Mpsi_best.resize(basePckr->workSize,basePckr->workSize);
    }))
    #ifdef TIMING
    basePckr->timer.toc(basePckr->TIMING_A9);
    #endif

    CudaGlobalPtr<CUDACOMPLEX> d_Fmic(allocator);
    CudaGlobalPtr<XFLOAT> d_Mmean(allocator);
    CudaGlobalPtr<XFLOAT> d_Mstddev(allocator);

    #ifdef TIMING
    basePckr->timer.tic(basePckr->TIMING_B1);
    #endif
    RFLOAT normfft = (RFLOAT) (basePckr->micrograph_size * basePckr->micrograph_size) / (RFLOAT) basePckr->nr_pixels_circular_mask;
    if (basePckr->do_read_fom_maps) {
        CTICTOC(timer, "readFromFomMaps_0", ({
        FileName fn_tmp = basePckr->getOutputRootName(fn_mic) + "_" + basePckr->fn_out + "_stddevNoise.spi";
        Image<RFLOAT> It;
        It.read(fn_tmp);
        Mstddev = It();
        }))
    } else {
        /*
         * Squared difference FOM:
         * Sum ( (X-mu)/sig  - A )^2 =
         *  = Sum((X-mu)/sig)^2 - 2 Sum (A*(X-mu)/sig) + Sum(A)^2
         *  = (1/sig^2)*Sum(X^2) - (2*mu/sig^2)*Sum(X) + (mu^2/sig^2)*Sum(1) - (2/sig)*Sum(AX) + (2*mu/sig)*Sum(A) + Sum(A^2)
         *
         * However, the squared difference with an "empty" ie all-zero reference is:
         * Sum ( (X-mu)/sig)^2
         *
         * The ratio of the probabilities thereby becomes:
         * P(ref) = 1/sqrt(2pi) * exp (( (X-mu)/sig  - A )^2 / -2 )   // assuming sigma = 1!
         * P(zero) = 1/sqrt(2pi) * exp (( (X-mu)/sig )^2 / -2 )
         *
         * P(ref)/P(zero) = exp(( (X-mu)/sig  - A )^2 / -2) / exp ( ( (X-mu)/sig )^2 / -2)
         *                = exp( (- (2/sig)*Sum(AX) + (2*mu/sig)*Sum(A) + Sum(A^2)) / - 2 )
         *
         *                Therefore, I do not need to calculate (X-mu)/sig beforehand!!!
         *
         */

        CTICTOC(timer, "Imic_insert", ({
        for (int i = 0; i < Imic().size(); i++)
            micTransformer.reals[i] = (XFLOAT) Imic().data[i];
        micTransformer.reals.cp_to_device();
        }))

        CTICTOC(timer, "runCenterFFT_0", ({
        runCenterFFT(micTransformer.reals, micTransformer.xSize, micTransformer.ySize, true, 1);
        }))

        CTICTOC(timer, "FourierTransform_0", ({
        micTransformer.forward();
        int FMultiBsize = ceilf((float) micTransformer.fouriers.getSize() * 2 / (float) BLOCK_SIZE);
        cuda_kernel_multi<<<FMultiBsize, BLOCK_SIZE>>>(
            (XFLOAT*) ~micTransformer.fouriers,
            (XFLOAT) 1 / (XFLOAT) micTransformer.reals.getSize(),
            micTransformer.fouriers.getSize() * 2
        );
        LAUNCH_HANDLE_ERROR(hipGetLastError());
        }))

        if (basePckr->highpass > 0.0) {
            CTICTOC(timer, "highpass", ({
            micTransformer.fouriers.streamSync();
            lowPassFilterMapGPU(
                micTransformer.fouriers,
                (size_t) 1,
                micTransformer.yFSize, micTransformer.xFSize,
                XSIZE(Imic()),
                basePckr->lowpass, basePckr->highpass,
                basePckr->angpix,
                2,
                true // false = lowpass, true = highpass
            );
            micTransformer.fouriers.streamSync();
            micTransformer.backward();
            micTransformer.reals.streamSync();
            }))
        }

        CTICTOC(timer, "F_cp", ({
        CudaGlobalPtr<CUDACOMPLEX> Ftmp(allocator);
        Ftmp.setSize(micTransformer.fouriers.getSize());
        Ftmp.device_alloc();
        micTransformer.fouriers.cp_on_device(Ftmp);
        }))

        // Also calculate the FFT of the squared micrograph
        CTICTOC(timer, "SquareImic", ({

        cuda_kernel_square<<<FMultiBsize, BLOCK_SIZE>>>(
            ~micTransformer.reals, micTransformer.reals.getSize()
        );
        LAUNCH_HANDLE_ERROR(hipGetLastError());
        }))

        CTICTOC(timer, "FourierTransform_1", ({

        micTransformer.forward();
        cuda_kernel_multi<<<FMultiBsize, BLOCK_SIZE>>>(
            (XFLOAT*) ~micTransformer.fouriers,
            (XFLOAT) 1 / (XFLOAT) micTransformer.reals.getSize(),
            micTransformer.fouriers.getSize() * 2
        );
        LAUNCH_HANDLE_ERROR(hipGetLastError());
        }))

        // The following calculate mu and sig under the solvent area at every position in the micrograph
        CTICTOC(timer, "calculateStddevAndMeanUnderMask", ({

        d_Mstddev.device_alloc(basePckr->workSize * basePckr->workSize);
        d_Mmean.device_alloc(basePckr->workSize * basePckr->workSize);


        /// TODO: Do this only once further up in scope
        CudaGlobalPtr<CUDACOMPLEX> d_Fmsk(basePckr->Finvmsk.size(), allocator);
        for (int i = 0; i< d_Fmsk.size ; i++) {
            d_Fmsk[i].x = basePckr->Finvmsk.data[i].real;
            d_Fmsk[i].y = basePckr->Finvmsk.data[i].imag;
        }
        d_Fmsk.put_on_device();
        d_Fmsk.streamSync();

        calculateStddevAndMeanUnderMask(
            Ftmp, micTransformer.fouriers,
            d_Fmsk, basePckr->nr_pixels_circular_invmask,
            d_Mstddev, d_Mmean,
            micTransformer.xFSize, micTransformer.yFSize,
            basePckr->micrograph_size, basePckr->workSize
        );


        /// TODO: remove this
        d_Mstddev.host_alloc();
        d_Mstddev.cp_to_host();
        d_Mstddev.streamSync();

        Mstddev.resizeNoCp(basePckr->workSize, basePckr->workSize);

        /// TODO: put this in a kernel
        for (int i = 0; i < d_Mstddev.size ; i ++) {
            Mstddev.data[i] = d_Mstddev[i];
            if (d_Mstddev[i] > (XFLOAT) 1E-10) {
                d_Mstddev[i] = 1 / d_Mstddev[i];
            } else {
                d_Mstddev[i] = 1;
            }
        }

        d_Mstddev.cp_to_device();
        d_Mstddev.streamSync();

        }))

        // From now on use downsized Fmic, as the cross-correlation with the references can be done at lower resolution
        CTICTOC(timer, "windowFourierTransform_0", ({

        d_Fmic.setSize((basePckr->workSize / 2 + 1) * basePckr->workSize);
        d_Fmic.device_alloc();
        windowFourierTransform2(
            Ftmp, d_Fmic,
            basePckr->micrograph_size / 2 + 1, basePckr->micrograph_size, 1, // Input dimensions
            basePckr->workSize        / 2 + 1, basePckr->workSize,        1  // Output dimensions
        );
        }))

        if (basePckr->do_write_fom_maps) {
            CTICTOC(timer, "writeToFomMaps", ({
            // TMP output
            FileName fn_tmp = basePckr->getOutputRootName(fn_mic) + "_" + basePckr->fn_out + "_stddevNoise.spi";
            Image<RFLOAT> It;
            It() = Mstddev;
            It.write(fn_tmp);
            }))
        }
    }

    // Now start looking for the peaks of all references
    // Clear the output vector with all peaks
    std::vector<Peak> peaks;
    CTICTOC(timer, "initPeaks", ({ peaks.clear(); }))
    #ifdef TIMING
    basePckr->timer.toc(basePckr->TIMING_B1);
    #endif

    if (basePckr->autopick_helical_segments) {
        if (basePckr->do_read_fom_maps) {
            FileName fn_ccf = basePckr->getOutputRootName(fn_mic) + "_" + basePckr->fn_out + "_combinedCCF.spi";
            Image<RFLOAT> It_float;
            It_float.read(fn_ccf);
            Mccf_best_combined = It_float();

            FileName fn_class = basePckr->getOutputRootName(fn_mic) + "_" + basePckr->fn_out + "_combinedCLASS.spi";
            It_int.read(fn_class);
            Image<int> It_int;
            Mclass_best_combined = It_int();
        } else {
            Mccf_best_combined.clear();
            Mccf_best_combined.resize(basePckr->workSize, basePckr->workSize);
            Mccf_best_combined.initConstant(-99.0e99);

            Mclass_best_combined.clear();
            Mclass_best_combined.resize(basePckr->workSize, basePckr->workSize);
            Mclass_best_combined.initConstant(-1);
        }
    }

    CudaGlobalPtr<XFLOAT> d_ctf(Fctf.size(), allocator);
    if (basePckr->do_ctf) {
        for (int i = 0; i < d_ctf.size; i++)
            d_ctf[i] = Fctf.data[i];
        d_ctf.put_on_device();
    }

    for (int iref = 0; iref < basePckr->Mrefs.size(); iref++) {

        CTICTOC(timer, "OneReference", ({
        RFLOAT expected_Pratio; // the expectedFOM for this (ctf-corrected) reference
        if (basePckr->do_read_fom_maps) {
            #ifdef TIMING
            basePckr->timer.tic(basePckr->TIMING_B2);
            #endif
            if (!basePckr->autopick_helical_segments) {
                CTICTOC(timer, "readFromFomMaps", ({
                FileName fn_tmp;
                Image<RFLOAT> It;

                fn_tmp.compose(basePckr->getOutputRootName(fn_mic) + "_" + basePckr->fn_out + "_ref", iref, "_bestCCF.spi");
                It.read(fn_tmp);
                Mccf_best = It();
                expected_Pratio = It.MDMainHeader.getValue(EMDL::IMAGE_STATS_MAX);  // Retrieve expected_Pratio from the header of the image

                fn_tmp.compose(basePckr->getOutputRootName(fn_mic) + "_" + basePckr->fn_out + "_ref", iref, "_bestPSI.spi");
                It.read(fn_tmp);
                Mpsi_best = It();
                }))
            }
            #ifdef TIMING
            basePckr->timer.toc(basePckr->TIMING_B2);
            #endif
        } else {
            #ifdef TIMING
            basePckr->timer.tic(basePckr->TIMING_B3);
            #endif
            CTICTOC(timer, "mccfInit", ({
            deviceInitValue(d_Mccf_best, (XFLOAT) -LARGE_NUMBER);
            }))
            CudaProjectorKernel projKernel = CudaProjectorKernel::makeKernel(
                cudaProjectors[iref],
                (int) basePckr->workSize / 2 + 1,
                (int) basePckr->workSize,
                1, // Zdim, always 1 in autopicker.
                (int) basePckr->workSize / 2 + 1 -1 // ?!?
            );

            int FauxStride = (basePckr->workSize / 2 + 1) * basePckr->workSize;

            #ifdef TIMING
            basePckr->timer.tic(basePckr->TIMING_B4);
            #endif
            CTICTOC(timer, "SingleProjection", ({
            dim3 blocks((int) ceilf((float) FauxStride / (float) BLOCK_SIZE), 1);
            if (basePckr->do_ctf) {
                cuda_kernel_rotateAndCtf<<<blocks, BLOCK_SIZE>>>(
                    ~cudaTransformer1.fouriers, ~d_ctf, 0, projKernel, 0
                );
            } else {
                cuda_kernel_rotateOnly<<<blocks, BLOCK_SIZE>>>(
                    ~cudaTransformer1.fouriers, 0, projKernel, 0
                );
            }
            LAUNCH_HANDLE_ERROR(hipGetLastError());
            }))
            #ifdef TIMING
            basePckr->timer.toc(basePckr->TIMING_B4);
            #endif
            /*
            *    FIRST PSI WAS USED FOR PREP CALCS - THIS IS NOW A DEDICATED SECTION
            *    -------------------------------------------------------------------
            */

            CTICTOC(timer, "PREP_CALCS", ({

            #ifdef TIMING
            basePckr->timer.tic(basePckr->TIMING_B5);
            #endif
            // Sjors 20 April 2016: The calculation for sum_ref_under_circ_mask, etc below needs to be done on original micrograph_size!
            CTICTOC(timer, "windowFourierTransform_FP", ({
            windowFourierTransform2(
                cudaTransformer1.fouriers,
                micTransformer.fouriers,
                basePckr->workSize / 2 + 1,        basePckr->workSize,        1, // Input dimensions
                basePckr->micrograph_size / 2 + 1, basePckr->micrograph_size, 1  // Output dimensions
            );
            }))

            CTICTOC(timer, "inverseFourierTransform_FP", ({
            micTransformer.backward();
            }))

            CTICTOC(timer, "runCenterFFT_FP", ({
            runCenterFFT(
                micTransformer.reals,
                (int) micTransformer.xSize, (int) micTransformer.ySize,
                false, 1
            );
            }))

            micTransformer.reals.cp_to_host();

            Maux.resizeNoCp(basePckr->micrograph_size, basePckr->micrograph_size);

            micTransformer.reals.streamSync();
            for (int i = 0; i < micTransformer.reals.size ; i ++)
                Maux.data[i] = micTransformer.reals[i];

            CTICTOC(timer, "setXmippOrigin_FP_0", ({ Maux.setXmippOrigin(); })
            /// TODO: check whether I need CenterFFT(Maux, false)
            // Sjors 20 Apr 2016: checked, somehow not needed.

            sum_ref_under_circ_mask = 0.0;
            sum_ref2_under_circ_mask = 0.0;
            RFLOAT suma2 = 0.0;
            RFLOAT sumn = 1.0;

            MultidimArray<RFLOAT> Mctfref(basePckr->particle_size, basePckr->particle_size);
            CTICTOC(timer, "setXmippOrigin_FP_1", ({
            Mctfref.setXmippOrigin();
            }))

            CTICTOC(timer, "suma_FP", ({
            FOR_ALL_ELEMENTS_IN_ARRAY2D(Mctfref, i, j) {
                // only loop over smaller Mctfref, but take values from large Maux!
                if (i * i + j * j < basePckr->particle_radius2) {
                    const auto &x = Maux.elem(i, j);
                    suma2 += x * x;
                    suma2 += 2.0 * x * rnd_gaus(0.0, 1.0);
                    sum_ref_under_circ_mask += x;
                    sum_ref2_under_circ_mask += x * x;
                    sumn += 1.0;
                }
            }
            sum_ref_under_circ_mask /= sumn;
            sum_ref2_under_circ_mask /= sumn;
            expected_Pratio = exp(suma2 / (2.0 * sumn));
            }))

            }))

            CTICTOC(timer, "AllPsi", ({
            int startPsi = 0;
            // for all batches
            for (int psiIter = 0; psiIter < cudaTransformer1.batchIters; psiIter++) {
                // psi-batches for possible memory-limits

                CTICTOC(timer, "Projection", ({
                dim3 blocks((int) ceilf((float) FauxStride / (float) BLOCK_SIZE), cudaTransformer1.batchSize[psiIter]);
                if (basePckr->do_ctf) {
                    cuda_kernel_rotateAndCtf<<<blocks, BLOCK_SIZE>>>(
                        ~cudaTransformer1.fouriers, ~d_ctf,
                        radians(basePckr->psi_sampling),
                        projKernel, startPsi
                    );
                } else {
                    cuda_kernel_rotateOnly<<<blocks, BLOCK_SIZE>>>(
                        ~cudaTransformer1.fouriers,
                        radians(basePckr->psi_sampling),
                        projKernel, startPsi
                    );
                }
                LAUNCH_HANDLE_ERROR(hipGetLastError());
                }))

                // Now multiply template and micrograph to calculate the cross-correlation
                CTICTOC(timer, "convol", ({
                dim3 blocks2(
                    (int) ceilf((float) FauxStride / (float) BLOCK_SIZE),
                    cudaTransformer1.batchSize[psiIter]
                );
                cuda_kernel_batch_convol_A<<<blocks2, BLOCK_SIZE>>>(
                    cudaTransformer1.fouriers.d_ptr, d_Fmic.d_ptr, FauxStride
                );
                LAUNCH_HANDLE_ERROR(hipGetLastError());
                }))

                CTICTOC(timer, "CudaInverseFourierTransform_1", ({
                cudaTransformer1.backward();
                HANDLE_ERROR(hipDeviceSynchronize());
                }))


                CTICTOC(timer, "runCenterFFT_1", ({
                runCenterFFT(
                    cudaTransformer1.reals,
                    (int) cudaTransformer1.xSize,
                    (int) cudaTransformer1.ySize,
                    false,
                    cudaTransformer1.batchSize[psiIter]
                );
                }))
                // Calculate ratio of prabilities P(ref)/P(zero)
                // Keep track of the best values and their corresponding iref and psi
                // ------------------------------------------------------------------
                // So now we already had precalculated: Mdiff2 = 1/sig*Sum(X^2) - 2/sig*Sum(X) + mu^2/sig*Sum(1)
                // Still to do (per reference): - 2/sig*Sum(AX) + 2*mu/sig*Sum(A) + Sum(A^2)
                CTICTOC(timer, "probRatio", ({
                HANDLE_ERROR(hipDeviceSynchronize());
                dim3 PR_blocks(ceilf((float)(cudaTransformer1.reals.size/cudaTransformer1.batchSize[psiIter])/(float)PROBRATIO_BLOCK_SIZE));
                cuda_kernel_probRatio<<<PR_blocks, PROBRATIO_BLOCK_SIZE>>>(
                    d_Mccf_best.d_ptr, d_Mpsi_best.d_ptr,
                    cudaTransformer1.reals.d_ptr,
                    d_Mmean.d_ptr, d_Mstddev.d_ptr,
                    cudaTransformer1.reals.size / cudaTransformer1.batchSize[0],
                    (XFLOAT) -2 * normfft,
                    (XFLOAT) 2 * sum_ref_under_circ_mask,
                    (XFLOAT) sum_ref2_under_circ_mask,
                    (XFLOAT) expected_Pratio,
                    cudaTransformer1.batchSize[psiIter],
                    startPsi, Npsi
                );
                LAUNCH_HANDLE_ERROR(hipGetLastError());
                startPsi += cudaTransformer1.batchSize[psiIter];
                }))

            } // end for psi-batches
            }))
            #ifdef TIMING
            basePckr->timer.toc(basePckr->TIMING_B6);
            #endif

            #ifdef TIMING
            basePckr->timer.tic(basePckr->TIMING_B7);
            #endif
            CTICTOC(timer, "output", ({
            d_Mccf_best.cp_to_host();
            d_Mpsi_best.cp_to_host();
            d_Mccf_best.streamSync();
            for (int i = 0; i < Mccf_best.size(); i++) {
                Mccf_best.data[i] = d_Mccf_best[i];
                Mpsi_best.data[i] = d_Mpsi_best[i];
            }
            }))

            if (basePckr->do_write_fom_maps && !basePckr->autopick_helical_segments) {
                CTICTOC(timer, "writeFomMaps", ({
                // TMP output
                FileName fn_tmp;
                Image<RFLOAT> It;
                It() = Mccf_best;
                // Store expected_Pratio in the header of the image..
                It.MDMainHeader.setValue(EMDL::IMAGE_STATS_MAX, expected_Pratio);  // Store expected_Pratio in the header of the image
                fn_tmp.compose(basePckr->getOutputRootName(fn_mic) + "_" + basePckr->fn_out + "_ref", iref, "_bestCCF.spi");
                It.write(fn_tmp);

                It() = Mpsi_best;
                fn_tmp.compose(basePckr->getOutputRootName(fn_mic) + "_" + basePckr->fn_out + "_ref", iref, "_bestPSI.spi");
                It.write(fn_tmp);
                }))

            }
            #ifdef TIMING
            basePckr->timer.toc(basePckr->TIMING_B7);
            #endif
            #ifdef TIMING
            basePckr->timer.toc(basePckr->TIMING_B3);
            #endif
        }


        /// TODO: FIX HELICAL SEGMENTS SUPPORT
        if (basePckr->autopick_helical_segments) {
            if (!basePckr->do_read_fom_maps) {
                // Combine Mccf_best and Mpsi_best from all refs
                for (long int n = 0; n < Mccf_best.size(); n++) {
                    RFLOAT new_ccf = Mccf_best[n];
                    RFLOAT old_ccf = Mccf_best_combined[n];
                    if (new_ccf > old_ccf) {
                        Mccf_best_combined[n] = new_ccf;
                        Mclass_best_combined[n] = iref;
                    }
                }
            }
        } else {
            #ifdef TIMING
            basePckr->timer.tic(basePckr->TIMING_B8);
            #endif
            // Now that we have Mccf_best and Mpsi_best, get the peaks
            std::vector<Peak> my_ref_peaks;
            CTICTOC(timer, "setXmippOriginX3", ({
            Mstddev.setXmippOrigin();
            Mccf_best.setXmippOrigin();
            Mpsi_best.setXmippOrigin();
            }))

            CTICTOC(timer, "peakSearch", ({
            basePckr->peakSearch(Mccf_best, Mpsi_best, Mstddev, iref, my_skip_side, my_ref_peaks, scale);
            }))

            CTICTOC(timer, "peakPrune", ({
            basePckr->prunePeakClusters(my_ref_peaks, min_distance_pix, scale);
            }))

            CTICTOC(timer, "peakInsert", ({
            // append the peaks of this reference to all the other peaks
            peaks.insert(peaks.end(), my_ref_peaks.begin(), my_ref_peaks.end());
            }))

            #ifdef TIMING
            basePckr->timer.toc(basePckr->TIMING_B8);
            #endif
        }
        }))
    }

    if (basePckr->autopick_helical_segments) {
        RFLOAT thres = basePckr->min_fraction_expected_Pratio;
        int peak_r_min = 1;
        std::vector<ccfPeak> ccf_peak_list;
        std::vector<std::vector<ccfPeak>> tube_coord_list, tube_track_list;
        std::vector<RFLOAT> tube_len_list;
        MultidimArray<RFLOAT> Mccfplot;

        Mccf_best_combined.setXmippOrigin();
        Mclass_best_combined.setXmippOrigin();
        basePckr->pickCCFPeaks(
            Mccf_best_combined, Mclass_best_combined, thres, peak_r_min,
            (basePckr->particle_diameter / basePckr->angpix),
            ccf_peak_list, Mccfplot, my_skip_side, scale
        );
        basePckr->extractHelicalTubes(
            ccf_peak_list, tube_coord_list, tube_len_list, tube_track_list,
            basePckr->particle_diameter / basePckr->angpix, basePckr->helical_tube_curvature_factor_max,
            basePckr->min_particle_distance / basePckr->angpix,
            basePckr->helical_tube_diameter / basePckr->angpix, scale
        );
        basePckr->exportHelicalTubes(
            Mccf_best_combined, Mccfplot, Mclass_best_combined,
            tube_coord_list, tube_track_list, tube_len_list,
            fn_mic, basePckr->fn_out,
            basePckr->particle_diameter / basePckr->angpix,
            basePckr->helical_tube_length_min / basePckr->angpix,
            my_skip_side, scale
        );

        if (basePckr->do_write_fom_maps) {

            Image<RFLOAT> It_float;
            It_float() = Mccf_best_combined;
            It_float.write(basePckr->getOutputRootName(fn_mic) + "_" + basePckr->fn_out + "_combinedCCF.spi");

            Image<int> It_int;
            It_int() = Mclass_best_combined;
            It_int.write(basePckr->getOutputRootName(fn_mic) + "_" + basePckr->fn_out + "_combinedCLASS.spi");

        }

        if (basePckr->do_write_fom_maps || basePckr->do_read_fom_maps) {
            Image<RFLOAT> It;
            It() = Mccfplot;
            It.write(basePckr->getOutputRootName(fn_mic) + "_" + basePckr->fn_out + "_combinedPLOT.spi");
        }
    } else {
        #ifdef TIMING
        basePckr->timer.tic(basePckr->TIMING_B9);
        #endif
        // Now that we have done all references, prune the list again...
        CTICTOC(timer, "finalPeakPrune", ({
        basePckr->prunePeakClusters(peaks, min_distance_pix, scale);
        }))

        // And remove all too close neighbours
        basePckr->removeTooCloselyNeighbouringPeaks(peaks, min_distance_pix, scale);

        // Write out a STAR file with the coordinates
        MetaDataTable MDout;
        for (int ipeak = 0; ipeak < peaks.size(); ipeak++) {
            MDout.addObject();
            MDout.setValue(EMDL::IMAGE_COORD_X, (RFLOAT) peaks[ipeak].x / scale);
            MDout.setValue(EMDL::IMAGE_COORD_Y, (RFLOAT) peaks[ipeak].y / scale);
            MDout.setValue(EMDL::PARTICLE_CLASS,         peaks[ipeak].ref + 1); // start counting at 1
            MDout.setValue(EMDL::PARTICLE_AUTOPICK_FOM,  peaks[ipeak].fom);
            MDout.setValue(EMDL::ORIENT_PSI,             peaks[ipeak].psi);
        }
        FileName fn_tmp = basePckr->getOutputRootName(fn_mic) + "_" + basePckr->fn_out + ".star";
        MDout.write(fn_tmp);
        #ifdef TIMING
        basePckr->timer.toc(basePckr->TIMING_B9);
        #endif
    }
}
